//nvcc -run -arch=sm_61 cub_test.cu
#include <stdio.h>
#include<vector>
#include<iostream>
#include<hip/hip_runtime.h>
#include "hipcub/hipcub.hpp"

#define SIZE 4865*1216*4*8

int main()
{

  hipDeviceReset();
 hipSetDevice(0);
  int *v;
  int *ids;
  v = (int*)malloc(SIZE * sizeof(int));
  ids = (int*)malloc(SIZE * sizeof(int));

  for(int i = 0; i < SIZE; i++){
    v[i] = rand()%(SIZE*4) + 32;
    ids[i] = i;
  }

  std::cout << "keys before: ";
  for(int i=0;i<16;i++)
    std::cout << " " << v[i];
  std::cout << std::endl;
  //
  std::cout << "indexes before: ";
  for(int i=0;i<16;i++)
    std::cout << " " << ids[i];
  std::cout << std::endl;

  int *dids, *dv;
  void *dtemp;

  hipMalloc((void**) &dids, SIZE * sizeof(int));
  hipMalloc((void**) &dv, SIZE * sizeof(int));
  hipMalloc((void**) &dtemp, SIZE * 2 * sizeof(int));

  hipMemcpy(dids, ids, SIZE * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dv, v, SIZE * sizeof(int), hipMemcpyHostToDevice);

  void *dsize = NULL;
  size_t expected_size = 0;
  hipcub::DeviceRadixSort::SortPairs(
                        dsize,
                        expected_size,
                        (const int*)dv,
                        dv,
                        (const int*)dids,
                        dids,
                        SIZE,
                        0,
                        32,
                        0,
                        true);

  std::cout << "SIZE " << expected_size << std::endl;
hipGetLastError();
  hipcub::DeviceRadixSort::SortPairs(
                        dtemp,
                        expected_size, //size,
                        (const int*)dv,
                        dv,
                        (const int*)dids,
                        dids,
                        SIZE,
                        0,
                        32,
                        0,
                        true);
hipDeviceSynchronize();
hipGetLastError();

  std::cout << "----------------------------------\n";
  int *v2; // = {1,2,3,4,5,6,7,8,9,10,11,12};
  int *ids2;// = {5,1,3,1,2,4,9,8,7,2,3,3};
  v2 = (int*)malloc(SIZE * sizeof(int));
  ids2 = (int*)malloc(SIZE * sizeof(int));

  hipMemcpy(v2, dv, SIZE * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(ids2, dids, SIZE * sizeof(int), hipMemcpyDeviceToHost);

std::cout << "keys after: ";
  for(int i=0;i<16;i++)
    std::cout << " " << v2[i];
  std::cout << std::endl;
  //
std::cout << "indexes after: ";
  for(int i=0;i<16;i++)
    std::cout << " " << ids2[i];
  std::cout << std::endl;

free(v); free(ids); free(v2); free(ids2);

  hipFree(dids);
  hipFree(dv);
  hipFree(dtemp);
	return 0;
}
