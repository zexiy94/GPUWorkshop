
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "hiprand/hiprand_kernel.h"
//#include <windows.h>
#define SIZE 128*128
#define MAX_STATE_SIZE 256
#define BLOCK_SIZE 256

__global__ void setup_kernel (hiprandStateMRG32k3a* state, unsigned long seed )//(curandState * state, unsigned long seed )

{
    int gid = threadIdx.x%MAX_STATE_SIZE; //+ blockIdx.x * blockDim.x;
    hiprand_init ( seed, gid, 0, &state[gid] );
} 

__global__ void simple_setup(hiprandStateMRG32k3a* state, int size )// (curandState * state, int size )
{
    int gid = threadIdx.x%MAX_STATE_SIZE;//+ blockIdx.x * blockDim.x;
    if(gid < size)hiprand_init ( 1937, gid, 0, &state[gid] );
} 

__global__ void generate( float* randArray, hiprandStateMRG32k3a* globalState, int size) //( float* randArray, curandState* globalState, int size) 
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
	if(gid < size){
		int chosenState = (blockIdx.x * 11 * threadIdx.x)%gridDim.x ;
		//int chosenState = gid;
		hiprandStateMRG32k3a localState = globalState[chosenState];
		randArray[gid] = hiprand_uniform( &localState );
		//saving back the state
		//globalState[chosenState] = localState;
	}
}

__global__ void generateHQ( float* randArray, hiprandStateMRG32k3a* globalState, int size) //( float* randArray, curandState* globalState, int size) 
{
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
	if(gid < size){
		//int chosenState = (blockIdx.x * 11 * threadIdx.x)%gridDim.x ;
		int chosenState = gid;
		hiprandStateMRG32k3a localState = globalState[chosenState];
		randArray[gid] = hiprand_uniform( &localState );
		//saving back the state
		globalState[chosenState] = localState;//every thread needs its own state...
	}
}


__host__ void full_rand_setup(float* randArray, int rand_size){
	hiprandStateMRG32k3a *allStates;
	hipMalloc((void**) &allStates, rand_size*sizeof(hiprandStateMRG32k3a));
	simple_setup<<<(rand_size+255)/256,256>>>(allStates, rand_size);
	generate<<<(rand_size+255)/256,256>>>(randArray, allStates, rand_size);
	hipFree(allStates);
}
int main(int argc,char **argv)
{

 printf("Starting the execution!\n");
 printf("Size of hiprandStateMRG32k3a state: %d\n", sizeof(hiprandStateMRG32k3a));

 hiprandStateMRG32k3a *d_states; 
 float *d_randArray, *d_output, *h_randArray;
 
 h_randArray=(float*)malloc(SIZE * sizeof(float));

 hipEvent_t gstart,gstop;
 hipEventCreate(&gstart);
 hipEventCreate(&gstop); 

 int stateSize = MAX_STATE_SIZE;//((SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE);
/////////////////// STEP 1 - ALLOCATING DEVICE MEMORY /////////////////
 printf("Allocating device memory!\n");
 hipMalloc((void**) &d_states, stateSize * sizeof(hiprandStateMRG32k3a));
 hipMalloc((void**) &d_randArray, SIZE * sizeof(float));
 hipMalloc((void**) &d_output, SIZE * sizeof(float));
 
 
//////////// STEP 3 - SETTING UP LAUNCH PARAMETERS /////////////////////
 dim3 dimGrid = dim3((SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE,1, 1);
 dim3 stateGrid = dim3((stateSize + BLOCK_SIZE - 1)/BLOCK_SIZE,1, 1);
 dim3 dimBlock = dim3(BLOCK_SIZE,1,1);
 printf("starting the calculation!\n");
 
 
 hipEventRecord(gstart, 0);
////////////////// STEP 4 - LAUNCH THE KERNEL //////////////////////////
//printf("setting up the states\n");
 simple_setup<<<stateGrid, dimBlock>>>(d_states,stateSize);
 generate<<<dimGrid, dimBlock>>>(d_randArray, d_states,SIZE);
 
 hipEventRecord(gstop, 0);
 hipEventSynchronize(gstop); 
 float gpu_time; 
 hipEventElapsedTime(&gpu_time, gstart, gstop);
 printf("GPU version has finished, it took %f ms\n",gpu_time );
//copy memory to print
hipMemcpy( h_randArray, d_randArray, SIZE * sizeof(float), hipMemcpyDeviceToHost);
for(int i =0; i< SIZE; i++){
 	if(i%10 == 0) printf("\n");
	printf(" %f",  h_randArray[i]);
 }
  printf("\n");
///////////////// STEP 6 - DEALOCATE DEVICE MEMORY /////////////////////
 printf("Deallocating GPU memory \n");
 hipFree(d_states);
 hipFree(d_randArray);
 hipFree(d_output); 
 
 free( h_randArray);

 
 hipEventDestroy(gstart); //cleaning up a bit
 hipEventDestroy(gstop);
 
 printf("That's all folks!\n");

 return 0;
}