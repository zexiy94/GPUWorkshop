#include "hip/hip_runtime.h"
// nvcc -arch=sm_37 -run vectoradd_multiple_exercise.cu
#include "hip/hip_runtime.h"
#include <stdio.h>


__global__ void kernel_vector_multi_add(float *result_vec, float *vec_a, float *vec_b, int vector_size, int number_of_vectors){
  ////// YOUR CODE GOES HERE!////////////////////////////////////////////
  //step 1.. define the thread ID
  int gid = ;
  int local_id = ;
  //step 2.. make sure the thread works whithin the array bounds

	//step 3.. compute the vector sum


}
/////////////////////////////////Serial version/////////////////////////////////////////////
void host_vector_multi_add(float *result_vec, float *vec_a, float *vec_b, int vector_size, int number_of_vectors){

  for(int i = 0 ; i < number_of_vectors; i++){
    	for(int elem = 0; elem < vector_size; elem++){
    		  result_vec[i * vector_size + elem] = vec_a[i * vector_size + elem] + vec_b[elem];
    	}
  }
}
/////////////////////////////////Diagnostic routines/////////////////////////////////////////////
int check_equal_float_vec(float *vec1,float *vec2,int size){
	int numerrors = 0;
	float dist;
	float tolerance = 0.0001f;
	for(int i =0; i< size; i++){
	    dist = (vec1[i] - vec2[i]) * (vec1[i] - vec2[i]);
		if(dist > tolerance) numerrors++;
	}
	if(numerrors ==0)printf("Congratulations you have 0 errors!\n");
	if(numerrors >0)printf("Wrong results, you have %d errors!\n", numerrors);

	return numerrors;
}
//////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
float *d_vec_out, *d_vec_a, *d_vec_b;
float *h_vec_out_cpu, *h_vec_out_gpu, *h_vec_a, *h_vec_b;
printf("\nStarting program execution..\n\n");
int vec_size = 4096;
int number_of_vectors = 16192;

printf("Allocating and creating problem data..\n");
int vec_size_bytes = vec_size * sizeof(float);
int vec_multi_size_bytes = number_of_vectors * vec_size * sizeof(float);
//allocation of host memory
h_vec_out_cpu = (float*)malloc(vec_multi_size_bytes);
h_vec_out_gpu = (float*)malloc(vec_multi_size_bytes);
h_vec_a = (float*)malloc(vec_multi_size_bytes);
h_vec_b = (float*)malloc(vec_size_bytes);

for(int i =0; i < vec_size; i++){
	h_vec_b[i] = (rand()%15)/10.0f;
}

for(int i =0; i < number_of_vectors * vec_size; i++){
	h_vec_a[i] = (rand()%10)/10.0f;
}
 ////
 //------ Step 1: Allocate the memory-------
 printf("Allocating Device Memory..\n");
//.... ALLOCATE THE VECTORS

//------ Step 2: Copy Memory to the device-------
printf("Transfering data to the Device..\n");
//.... COPY THE VECTORS

//------ Step 3: Prepare launch parameters-------
printf("preparing launch parameters..\n");

dim3 dimGrid = dim3((, 1, 1);//.... CONFIGURE THE GRID IN BLOCKS OF 256 THREADS BLOCKS
dim3 dimBlock = dim3(256,1,1);
//------ Step 4: Launch device kernel-------
printf("Launch Device Kernel.\n");

// YOUR KERNEL LAUNCH GOES HERE------------------------>>>>>>>>>


//------ Step 5: Copy Memory back to the host-------
printf("Transfering result data to the Host..\n");
//.... COPY THE RESULT VECTOR

 //
 printf("CPU version...\n");
host_vector_multi_add(h_vec_out_cpu, h_vec_a, h_vec_b, vec_size , number_of_vectors);//serial version to compare
printf("Checking solutions..\n");
check_equal_float_vec(h_vec_out_gpu, h_vec_out_cpu, vec_size * number_of_vectors);
//

// -----------Step 6: Free the memory --------------
printf("Deallocating device memory..\n");
//.... FREE  THE VECTORS

free(h_vec_a);
free(h_vec_b);
free(h_vec_out_cpu);
free(h_vec_out_gpu);

return 0;
}

// hipEvent_t gstart, gstop;
// hipEventCreate(&gstart);
// hipEventCreate(&gstop);
