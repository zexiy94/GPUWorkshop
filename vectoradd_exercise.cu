#include "hip/hip_runtime.h"
// nvcc -arch=sm_37 -run vectoradd_exercise.cu
#include "hip/hip_runtime.h"
#include <stdio.h>


__global__ void kernel_vector_add(float *result_vec, float *vec_a, float *vec_b, int vector_size){
  ////// YOUR CODE GOES HERE!////////////////////////////////////////////
  //step 1.. define the thread ID

  //step 2.. make sure the thread works whithin the array bounds

	//step 3.. compute the vector sum

}
/////////////////////////////////Serial version/////////////////////////////////////////////
void host_vector_add(float *result_vec, float *vec_a, float *vec_b, int vector_size){

	for(int elem = 0; elem < vector_size; elem++){
		result_vec[elem] = vec_a[elem] + vec_b[elem];
	}

}
/////////////////////////////////Diagnostic routines/////////////////////////////////////////////
int check_equal_float_vec(float *vec1,float *vec2,int size){
	int numerrors = 0;
	float dist;
	float tolerance = 0.0001f;
	for(int i =0; i< size; i++){
	    dist = (vec1[i] - vec2[i]) * (vec1[i] - vec2[i]);
		if(dist > tolerance) numerrors++;
	}
	if(numerrors ==0)printf("Congratulations you have 0 errors!\n");
	if(numerrors >0)printf("Wrong results, you have %d errors!\n", numerrors);

	return numerrors;
}
//////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
float *d_vec_out, *d_vec_a, *d_vec_b;
float *h_vec_out_cpu, *h_vec_out_gpu, *h_vec_a, *h_vec_b;
printf("\nStarting program execution..\n\n");
int vec_size = 4096;

printf("Allocating and creating problem data..\n");
int vec_size_bytes = vec_size * sizeof(float);
//allocation of host memory
h_vec_out_cpu = (float*)malloc(vec_size_bytes);
h_vec_out_gpu = (float*)malloc(vec_size_bytes);
h_vec_a = (float*)malloc(vec_size_bytes);
h_vec_b = (float*)malloc(vec_size_bytes);

for(int i =0; i < vec_size; i++){
	h_vec_a[i] = (rand()%10)/10.0f;
	h_vec_b[i] = (rand()%15)/10.0f;
}
 ////
 //------ Step 1: Allocate the memory-------
 printf("Allocating Device Memory..\n");
hipMalloc((void**)&d_vec_out, vec_size_bytes);
//.... ALLOCATE THE REST OF THE VECTORS

//------ Step 2: Copy Memory to the device-------
printf("Transfering data to the Device..\n");
hipMemcpy(d_vec_a, h_vec_a, vec_size_bytes, hipMemcpyHostToDevice);
//.... COPY THE REST OF THE VECTORS

//------ Step 3: Prepare launch parameters-------
printf("preparing launch parameters..\n");

dim3 dimGrid = //.... CONFIGURE THE GRID IN BLOCKS OF 256 THREADS BLOCKS
dim3 dimBlock = dim3(256,1,1);
//------ Step 4: Launch device kernel-------
printf("Launch Device Kernel.\n");
// YOUR KERNEL LAUNCH GOES HERE------------------------>>>>>>>>>

//------ Step 5: Copy Memory back to the host-------
printf("Transfering result data to the Host..\n");
hipMemcpy(h_vec_out_gpu, d_vec_out, vec_size_bytes, hipMemcpyDeviceToHost);

 //
 printf("CPU version...\n");
host_vector_add(h_vec_out_cpu, h_vec_a, h_vec_b, vec_size);//serial version to compare
printf("Checking solutions..\n");
check_equal_float_vec(h_vec_out_gpu, h_vec_out_cpu, vec_size);
//

// -----------Step 6: Free the memory --------------
printf("Deallocating device memory..\n");
hipFree(d_vec_out);
//.... FREE THE REST OF THE VECTORS

free(h_vec_a);
free(h_vec_b);
free(h_vec_out_cpu);
free(h_vec_out_gpu);

return 0;
}

// hipEvent_t gstart, gstop;
// hipEventCreate(&gstart);
// hipEventCreate(&gstop);
