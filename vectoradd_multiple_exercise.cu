#include "hip/hip_runtime.h"
// nvcc -arch=sm_37 -run vectoradd_multiple_solution.cu cuda_helper.cu
#include "hip/hip_runtime.h"
#include "chTimer.h"
#include "cuda_helper.h"
#include <stdio.h>


__global__ void kernel_vector_multi_add(float *result_vec, float *vec_a, float *vec_b, int vector_size, int number_of_vectors){
  ////// YOUR CODE GOES HERE!////////////////////////////////////////////
  //step 1.. define the thread ID
  int gid = ;
  int local_id = ;//remember b is a single vector while a is a batch of them
  //step 2.. make sure the thread works whithin the array bounds

	//step 3.. compute the vector sum


}
/////////////////////////////////Serial version/////////////////////////////////////////////
void host_vector_multi_add(float *result_vec, float *vec_a, float *vec_b, int vector_size, int number_of_vectors){

  for(int i = 0 ; i < number_of_vectors; i++){
    	for(int elem = 0; elem < vector_size; elem++){
    		  result_vec[i * vector_size + elem] = vec_a[i * vector_size + elem] + vec_b[elem];
    	}
  }
}
/////////////////////////////////Diagnostic routines/////////////////////////////////////////////
int check_equal_float_vec(float *vec1,float *vec2,int size){
	int numerrors = 0;
	float dist;
	float tolerance = 0.0001f;
	for(int i =0; i< size; i++){
	    dist = (vec1[i] - vec2[i]) * (vec1[i] - vec2[i]);
		if(dist > tolerance) numerrors++;
	}
	if(numerrors ==0)printf("Congratulations you have 0 errors!\n");
	if(numerrors >0)printf("Wrong results, you have %d errors!\n", numerrors);

	return numerrors;
}
//////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
cpuClock cpuck;
cudaClock ck;
float *d_vec_out, *d_vec_a, *d_vec_b;
float *h_vec_out_cpu, *h_vec_out_gpu, *h_vec_a, *h_vec_b;
printf("\nStarting program execution..\n\n");
int vec_size = 4096;
int number_of_vectors = 16192;

printf("Allocating and creating problem data..\n");
int vec_size_bytes = vec_size * sizeof(float);
int vec_multi_size_bytes = number_of_vectors * vec_size * sizeof(float);
//allocation of host memory
h_vec_out_cpu = (float*)malloc(vec_multi_size_bytes);
h_vec_out_gpu = (float*)malloc(vec_multi_size_bytes);
h_vec_a = (float*)malloc(vec_multi_size_bytes);
h_vec_b = (float*)malloc(vec_size_bytes);

for(int i =0; i < vec_size; i++){
	h_vec_b[i] = (rand()%15)/10.0f;
}

for(int i =0; i < number_of_vectors * vec_size; i++){
	h_vec_a[i] = (rand()%10)/10.0f;
}
 ////
 //------ Step 1: Allocate the memory-------
 printf("Allocating Device Memory..\n");


//------ Step 2: Copy Memory to the device-------
printf("Transfering data to the Device..\n");


//------ Step 3: Prepare launch parameters-------
printf("preparing launch parameters..\n");

//------ Step 4: Launch device kernel-------
printf("Launch Device Kernel.\n");

// YOUR KERNEL LAUNCH GOES HERE------------------------>>>>>>>>>
cudaTick(&ck);

cudaTock(&ck, "kernel_vector_multi_add");

//------ Step 5: Copy Memory back to the host-------
printf("Transfering result data to the Host..\n");


 //
 printf("CPU version...\n");
 cpuTick(&cpuck);
host_vector_multi_add(h_vec_out_cpu, h_vec_a, h_vec_b, vec_size , number_of_vectors);//serial version to compare
cpuTock(&cpuck, "host_vector_multi_add");
std::cout << "the gpu is " << cpuck.elapsedMicroseconds/ck.elapsedMicroseconds << " times faster" << std::endl;
printf("Checking solutions..\n");
check_equal_float_vec(h_vec_out_gpu, h_vec_out_cpu, vec_size * number_of_vectors);
//

// -----------Step 6: Free the memory --------------
printf("Deallocating device memory..\n");

free(h_vec_a);
free(h_vec_b);
free(h_vec_out_cpu);
free(h_vec_out_gpu);

return 42;
}

// hipEvent_t gstart, gstop;
// hipEventCreate(&gstart);
// hipEventCreate(&gstop);
